#include "hip/hip_runtime.h"
#include "vectorAddJob.h"
#include <cmath>

void CUDART_CB VectorAddJob::vectorAddKernelCallback(hipStream_t stream,
                                                     hipError_t status,
                                                     void *data) {

  // get the kernel launch config that has to be cleaned up and potentially
  // checked for correctness.
  auto *kernelInfo = static_cast<KernelLaunchInfoBase *>(data);

  // push the job to the clean up queue which the scheduler will handle in its
  // own thread.
  // current time is called inside the cuda runtime thread spawned by the
  // callback => safe to call host function because it will not interfere with
  // the main thread.
  float currentTime = getCurrentTime();
  CompletionQueue::getCompletionQueue().push(
      {kernelInfo->jobPtr, currentTime, kernelInfo});
}

// callback constructor.
void VectorAddJob::addVectorAddKernelCallback(Job *job, hipStream_t stream,
                                              float *d_A, float *d_B,
                                              float *d_C, float *A, float *B,
                                              float *C) {

  VectorAddJobLaunchInfo *kernelInfo = new VectorAddJobLaunchInfo(job);
  // register the callback for the given stream.
  hipStreamAddCallback(stream, vectorAddKernelCallback, kernelInfo, 0);
}

void VectorAddJob::execute() {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> realDist(1.0, 100.0);

  // set the stream's mask using libsmctrl.
  if (!this->TPCMasks.empty()) {
    uint64_t mask = this->combineMasks();
    libsmctrl_set_stream_mask((void *)(this->kernelStream), mask);
  }

  // fill up two arrays with values.
  for (int i = 0; i < this->vectorSize; i++) {
    A[i] = realDist(gen);
    B[i] = realDist(gen);
  }

  // copy the contents of the host arrays to the device arrays in an async way
  // before the kernel is launched.
  hipMemcpyAsync(d_A, A, this->vectorSize, hipMemcpyHostToDevice,
                  kernelStream);
  hipMemcpyAsync(d_B, B, this->vectorSize, hipMemcpyHostToDevice,
                  kernelStream);

  // kernel launch.
  vectorAddKernel<<<this->threadBlocks, this->threadsPerBlock, 0,
                    kernelStream>>>(d_A, d_B, d_C, this->vectorSize);

  //  copy the result back into the host array.
  hipMemcpyAsync(C, d_C, this->vectorSize, hipMemcpyDeviceToHost,
                  kernelStream);
  addVectorAddKernelCallback(this, kernelStream, d_A, d_B, d_C, A, B, C);
}

VectorAddJob::VectorAddJob(int threadsPerBlock, int vectorSize) {

  this->threadsPerBlock = threadsPerBlock;
  this->vectorSize = vectorSize;
  this->nrOfBytes = this->vectorSize * sizeof(float);

  // kernel launch config.
  hipMalloc(&d_A, this->nrOfBytes);
  hipMalloc(&d_B, this->nrOfBytes);
  hipMalloc(&d_C, this->nrOfBytes);

  hipStreamCreate(&kernelStream);

  hipHostAlloc((void **)&A, this->nrOfBytes, hipHostMallocDefault);
  hipHostAlloc((void **)&B, this->nrOfBytes, hipHostMallocDefault);
  hipHostAlloc((void **)&C, this->nrOfBytes, hipHostMallocDefault);

  // in this case the total amount of threads is the same as the size of the
  // vector because each thread will calculate one addition.
  int totalThreads = vectorSize;
  this->threadBlocks =
      std::ceil(totalThreads + this->threadsPerBlock - 1 / threadsPerBlock);
  int neededSMs =
      ceil((float)totalThreads /
           (float)DeviceInfo::getDeviceProps()->getMaxThreadsPerSM());

  if (neededSMs < 1) {
    this->neededTPCs = 1;
    return;
  }
  this->neededTPCs = ceil((float)neededSMs /
                          (float)DeviceInfo::getDeviceProps()->getSMsPerTPC());
}

std::string VectorAddJob::getMessage() { return "vector addition done\n"; }

VectorAddJob::~VectorAddJob() {
  hipStreamSynchronize(this->kernelStream);

  hipStreamDestroy(this->kernelStream);

  // Clean up other resources
  hipFree(this->d_A);
  hipFree(this->d_B);
  hipFree(this->d_C);
  hipHostFree(this->A);
  hipHostFree(this->B);
  hipHostFree(this->C);
}
