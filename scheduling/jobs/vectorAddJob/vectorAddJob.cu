#include "hip/hip_runtime.h"
#include "vectorAddJob.h"
#include <cmath>

void CUDART_CB VectorAddJob::vectorAddKernelCallback(hipStream_t stream,
                                                     hipError_t status,
                                                     void *data) {

  // get the kernel launch config that has to be cleaned up and potentially
  // checked for correctness.
  auto *kernelInfo = static_cast<KernelLaunchInfoBase *>(data);

  // push the job to the clean up queue which the scheduler will handle in its
  // own thread.
  // current time is called inside the cuda runtime thread spawned by the
  // callback => safe to call host function because it will not interfere with
  // the main thread.
  float currentTime = getCurrentTime();
  CompletionQueue::getCompletionQueue().push(
      {kernelInfo->jobPtr, currentTime, kernelInfo});
}

// callback constructor.
void VectorAddJob::addVectorAddKernelCallback(Job *job, hipStream_t stream,
                                              float *d_A, float *d_B,
                                              float *d_C, float *A, float *B,
                                              float *C) {

  VectorAddJobLaunchInfo *kernelInfo = new VectorAddJobLaunchInfo(job);
  // register the callback for the given stream.
  hipStreamAddCallback(stream, vectorAddKernelCallback, kernelInfo, 0);
}

void VectorAddJob::execute() {

  // set the stream's mask using libsmctrl.
  if (!this->TPCMasks.empty()) {
    uint64_t mask = this->combineMasks();
    libsmctrl_set_stream_mask((void *)(this->kernelStream), mask);
  }

  // fill up two arrays with values.
  for (int i = 0; i < this->vectorSize; i++) {
    A[i] = i;
    B[i] = i + i;
  }

  // copy the contents of the host arrays to the device arrays in an async way
  // before the kernel is launched.
  hipMemcpyAsync(d_A, A, nrOfElements, hipMemcpyHostToDevice, kernelStream);
  hipMemcpyAsync(d_B, B, nrOfElements, hipMemcpyHostToDevice, kernelStream);

  // kernel launch.
  vectorAddKernel<<<this->threadBlocks, this->threadsPerBlock, 0,
                    kernelStream>>>(d_A, d_B, d_C, this->vectorSize);

  //  copy the result back into the host array.
  hipMemcpyAsync(C, d_C, nrOfElements, hipMemcpyDeviceToHost, kernelStream);
  addVectorAddKernelCallback(this, kernelStream, d_A, d_B, d_C, A, B, C);
}

VectorAddJob::VectorAddJob(int threadsPerBlock, int vectorSize) {

  this->threadsPerBlock = threadsPerBlock;
  this->vectorSize = vectorSize;

  // kernel launch config.
  hipMalloc(&d_A, this->vectorSize * sizeof(float));
  hipMalloc(&d_B, this->vectorSize * sizeof(float));
  hipMalloc(&d_C, this->vectorSize * sizeof(float));

  hipStreamCreate(&kernelStream);

  nrOfElements = this->vectorSize * sizeof(float);
  hipHostAlloc((void **)&A, nrOfElements, hipHostMallocDefault);
  hipHostAlloc((void **)&B, nrOfElements, hipHostMallocDefault);
  hipHostAlloc((void **)&C, nrOfElements, hipHostMallocDefault);

  // in this case the total amount of threads is the same as the size of the
  // vector because each thread will calculate one addition.
  int totalThreads = vectorSize;
  this->threadBlocks =
      std::ceil(totalThreads + this->threadsPerBlock - 1 / threadsPerBlock);
  int neededSMs =
      ceil((float)totalThreads /
           (float)DeviceInfo::getDeviceProps()->getMaxThreadsPerSM());

  if (neededSMs < 1) {
    this->neededTPCs = 1;
    return;
  }
  this->neededTPCs =
      ceil(neededSMs / DeviceInfo::getDeviceProps()->getSMsPerTPC());
}

std::string VectorAddJob::getMessage() { return "vector addition done\n"; }
