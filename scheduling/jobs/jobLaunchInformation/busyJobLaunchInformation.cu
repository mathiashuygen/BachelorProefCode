#include "busyJobLaunchInformation.h"

void BusyJobLaunchInfo::cleanup() {
  hipStreamSynchronize(this->kernelStream);

  hipHostFree(this->hostPtr);
  hipFree(this->devicePtr);
  hipFree(this->timerptr);
  hipStreamDestroy(this->kernelStream);
}
