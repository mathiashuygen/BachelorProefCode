#include "printJobLaunchInformation.h"
#include <iostream>
void PrintJobLaunchInfo::cleanup() {
  std::cout << "delete print job launch info\n";
  hipStreamSynchronize(kernelStream);
  hipHostFree(hostPtr);
  hipFree(devicePtr);
  hipStreamDestroy(kernelStream);
}
